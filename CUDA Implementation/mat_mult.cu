#include "hip/hip_runtime.h"
/** 
* Program for facilitating Matrix Multiplication
*/


/**
 * Header files that are used
 */
#include <stdio.h>
#include <stdlib.h>

/**
 * Preprocessor directives
 */
#define EXEC_CPU     0
#define VECTOR_SIZE  1000000000
#define ROWS         3000
#define K            4000
#define COLS         5000
#define BLOCK_SIZE   32 
#define INTSIZE sizeof(unsigned int)

/**
 * Kernel for matrix multiplication (GPU)
 */
__global__ void matMult(int* a, int* b, int* res, unsigned  int rows, unsigned int k, unsigned int cols)
{
    /**
     * Getiing the x and Y dimension iterators
     */
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;

    /**
     * Sum being set to 0
     */
    unsigned int sum = 0;

    if(r< rows && c< cols)
    {
        for(int x=0; x<k; x++)
        {
            sum += a[r*k +x] + b[x*cols + c]; 
        }
            /**
              * Storing the result
              */
        res[r*cols + c] = sum;
    }
}

int main( void ) 
{ 

    /** 
     * Set device that we will use for our cuda code
     * It will be either 0 or 1
     */
    hipSetDevice(1);

    /**
     * Seeding the randomness
     */
    srand(time(NULL));
    
    /** 
     * Time Variables
     */
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    /**
     * Express matrix elements as 1 dimension
     */
    unsigned int aSize =  ROWS * K * INTSIZE;
    unsigned int bSize =  K * COLS* INTSIZE;
    unsigned int cSize =  ROWS * COLS * INTSIZE;

    int *a, *b, *c_cpu, *c_gpu;

    /**
     * Allocating memory on the Host (CPU)
     */
    hipHostMalloc((void**)&a,aSize);
    hipHostMalloc((void**)&b,bSize);
    hipHostMalloc((void**)&c_cpu,cSize);
    hipHostMalloc((void**)&c_gpu,cSize);
 
    /** 
     * Pointers in GPU memory
     */
    int *dev_a;
    int *dev_b;
    int *dev_c;

    /** 
     * Fill the arrays 'a' and 'b' on the CPU
     */
    for(int r=0; r<ROWS; r++)
    {
        for(int c=0; c<K; c++)
        {
            a[ r*K + c] = rand()%10;
        }
    }

    for(int r=0; r<K; r++)
    {
        for(int c=0; c<COLS; c++)
        {
            b[ r*COLS + c ] = rand()%10;
        }
    }

    /** 
     * CPU Calculation
     */
    printf("Running sequential job.\n");

    /**
     * Starting the timer
     */
    hipEventRecord(start,0);

    if(EXEC_CPU)
    {
        /** 
         * Calculate C in the CPU
         */
        for(unsigned int r=0; r<ROWS; r++)
        {
            for(unsigned int c=0; c<COLS; c++)
            {

                int sum = 0; 
                for(int k=0; k<K;k++)
                {
                    sum +=  a[r*K + k] + b[k*COLS + c];
                }

                c_cpu[r*COLS + c] = sum;
            }
        }
    }

    /**
     * Stopping the timer
     */
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    /**
     * Time elpased
     */
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);

    /** 
     * Allocate the memory on the GPU
     */
    hipMalloc( (void**)&dev_a,  aSize);
    hipMalloc( (void**)&dev_b,  bSize);
    hipMalloc( (void**)&dev_c,  cSize);

    /** 
     * Copy the arrays 'a' and 'b' to the GPU
     */
    hipMemcpy( dev_a, a, aSize, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, bSize, hipMemcpyHostToDevice );

    /** 
     * GPU Calculation
     */
    printf("Running parallel job.\n");

    unsigned int gridRows =  (ROWS + BLOCK_SIZE - 1)/ BLOCK_SIZE; 
    unsigned int gridCols =  (COLS+ BLOCK_SIZE - 1)/ BLOCK_SIZE; 

    /**
     * Grid(s) and Block(s) division
     */
    dim3 grids(gridCols, gridRows);
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);

    /**
     * Starting the timer
     */
    hipEventRecord(start,0);
    
    /**
     * Summon the Kernel
     */
    matMult<<<grids, blocks>>>(dev_a, dev_b, dev_c, ROWS, K, COLS);

    /**
     * Ending the timer
     */
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    /**
     * Time taken
     */
    hipEventElapsedTime(&time, start, stop);
    printf("\tParallel Job Time: %.2f ms\n", time);

    /**
     * Getting the result(s) back
     */
    hipMemcpy( c_gpu, dev_c, cSize, hipMemcpyDeviceToHost);

    if(EXEC_CPU){
        /** 
         * Compare the results
         */
        int error = 0;
        for(unsigned int r=0; r<ROWS; r++)
        {
            for(unsigned int c=0; c<COLS; c++)
            {
                if (c_cpu[r*COLS + c] != c_gpu[r*COLS + c]){
                    error = 1;
                    break;
                }
            }
        }

        if (error == 0)
        {
            printf ("Correct result. No errors were found.\n");

        }
    }

    /**
     * Freeing the memory
     */
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c_cpu);
    hipHostFree(c_gpu);

    return 0;
}